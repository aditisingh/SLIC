#include <fstream>
#include <iostream>
#include <stdio.h>
#include <string>
#include <sstream>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <ctime>
#include <vector>
#include <cstdlib>
#include <algorithm>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>


using namespace std;

//handlerror declaration : to display file and line numbers of erroneous lines
static void HandleError( hipError_t err, const char *file, int line ) {
	if (err != hipSuccess) {
		cout<<hipGetErrorString(err)<<" in "<< file <<" at line "<< line<<endl;	//this will print the line and filename too
	}
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))//macro 

#define min(a, b) ((a) > (b))? (b): (a) //macro for min, to use in kernels
#define max(a, b) ((a) > (b))? (a): (b)//macro for max, to use in kernels

// storing RGB values for rgb colorspace images
struct pixel_RGB
{
	unsigned char r;  //Red values
	unsigned char g;  //Green values
	unsigned char b;  //Blue Values
};

// storing values for xyz and lab colorspace images
struct pixel_XYZ
{
	float x;  //X for XYZ colorspace, L for LAB colorspace
	float y;  //Y for XYZ colorspace, A for LAB colorspace
	float z;  //Z for XYZ colorspace, B for LAB colorspace
};

//store coordinates for each cluster centres
struct point
{ 
	int x;  //x-ccordinate
	int y;  //y-coordinate
};

__global__ void RGB2LAB(pixel_RGB* img, int img_wd, int img_ht, pixel_XYZ* LAB_img)
{
	unsigned int c=blockIdx.x*blockDim.x + threadIdx.x;	//row value using x-index of current thread
	unsigned int r=blockIdx.y*blockDim.y + threadIdx.y;	//column value using y-index of current thread
	unsigned int idx=r*img_wd+c;	//row major index

	if(idx>img_wd*img_ht)	//degenerate values
		return;

	 //read the RGB channel values
    int R=img[idx].r; 
    int G=img[idx].g;
    int B=img[idx].b;

    //normalize these values
    double var_R=double(R)/255;
    double var_G=double(G)/255;
    double var_B=double(B)/255;

    //linearize it to give XYZ colorspace
    double X = var_R * 0.4124 + var_G * 0.3576 + var_B * 0.1805;
    double Y = var_R * 0.2126 + var_G * 0.7152 + var_B * 0.0722;
    double Z = var_R * 0.0193 + var_G * 0.1192 + var_B * 0.9505;

    //Normalize XYZ values
    X=X/0.95047;
    Y=Y/1.00000;
    Z=Z/1.088969;

    //Conversion of XYZ to LAB Values
    double Y3=pow(Y,1/3);

    double T=0.008856;//threshold
    double fx=(X>T)?(pow(X,double(1)/3)):(7.787*X+(16/116));
    double fy=(Y>T)?(pow(Y,double(1)/3)):(7.787*Y+(16/116));
    double fz=(Z>T)?(pow(Z,double(1)/3)):(7.787*Z+(16/116));


    double L=(Y>T)?(116*Y3 - 16):(903.3*Y);
    double a = 500 * (fx - fy);
    double b = 200 * (fy - fz);

    //saving the calculations to image
    LAB_img[idx].x=L;
    LAB_img[idx].y=a;
    LAB_img[idx].z=b;

}

int min_index(float* array, int size, int x1, int x2, int y1, int y2, int img_wd) //find the index of min value a given region
{
  int index=(x1+1)+(y1+1)*img_wd; //initialize to the centre index
  for(int x=x1;x<x2;x++)
  {
    for(int y=y1;y<y2;y++)
    {
      if(array[y*img_wd+x]<array[index])
        index=y*img_wd+x;
    }
  }
  return index;
}

__global__ void label_assignment(int* labels_gpu, pixel_XYZ* Pixel_LAB_gpu, point* centers_gpu, int S, int img_wd, int img_ht, int m, float* d_gpu, int k1)
{

	unsigned int index = blockIdx.x*blockDim.x+ threadIdx.x; //find threadindex of cluster center
	if(index>=k1) //for degenerate cases
		return;//}	

	// // finding centre coordinates
	int x_center=centers_gpu[index].x;//find x coordinate of the cluster centre
	int y_center=centers_gpu[index].y;//find y coordinate of the cluster centre

	int centre_idx=y_center*img_wd+x_center;//find index in image row major form

	int L_x1=max(0, x_center-S), L_x2=min(x_center+S,img_wd-1),L_y1= max(0,y_center-S),L_y2= min(y_center+S, img_ht-1);

	for(int x_coord=L_x1;x_coord<=L_x2;x_coord++) //look in 2S x 2S neighborhood
	{//taking care it doesn't go out of the image
		for(int y_coord=L_y1;y_coord<=L_y2;y_coord++)
	 	{

			int j=y_coord*img_wd+x_coord; // find global index of the pixel

			float d_c = sqrt(pow((Pixel_LAB_gpu[centre_idx].x-Pixel_LAB_gpu[j].x),2) + pow((Pixel_LAB_gpu[centre_idx].y-Pixel_LAB_gpu[j].y),2) + pow((Pixel_LAB_gpu[centre_idx].z-Pixel_LAB_gpu[j].z),2)); //color proximity;
			float d_s = sqrtf((x_coord-x_center)*(x_coord-x_center)+(y_coord-y_center)*(y_coord-y_center)); //spatial proximity
			float D=powf(powf(d_c,2)+powf(m*d_s/S,2),0.5);//effective distance
             //if it is lesser than current distance, update
			
			if(D<d_gpu[j])
			 {
				d_gpu[j]=D;//store new center
				labels_gpu[j]=index;//label as the number of cluster centre
			}

		}
	}
}

__global__ void update_centres(int* labels_gpu, point* centers_gpu, int S, int img_wd, int img_ht, int k1)
{
	size_t index = blockIdx.x*blockDim.x+ threadIdx.x; //thread index
	
	if(index>=k1)
		return;

	// finding centre coordinates
	int centre_x=centers_gpu[index].x;//find x-coordinate of the centre
	int centre_y=centers_gpu[index].y;//find y-coordinate of the centre
	
	//finding the label of cluster, this will be center's label
	int i=labels_gpu[centre_y*img_wd+centre_x]; //finding the label of centre

	int x_mean=0, y_mean=0, count=0;//mean will store update cluster
	int L_x1=max(0, centre_x-S), L_x2=min(centre_x+S,img_wd-1),L_y1= max(0,centre_y-S),L_y2= min(centre_y+S, img_ht-1);
	//storing 2Sx2S coordinate ranges from(L_x1,L_y1) to (L_x2,L_y2)

	for(int x_coord=L_x1;x_coord<=L_x2;x_coord++) //look in 2S x 2S neighborhood
	{
		for(int y_coord=L_y1;y_coord<=L_y2;y_coord++)
	 	{
			int pt_idx=y_coord*img_wd+x_coord;//index of the points in the neighborhood

			if(labels_gpu[pt_idx]==i)//if the label is the cluster centres, add x and y coordinates to x_mean and y_mean
			{
				x_mean+=x_coord; 
				y_mean+=y_coord;
				count++;//increment the counter
			}
		}
	}
	if(count)//if any counts
	{
		centers_gpu[index].x=x_mean/count;//calculate mean x and y coordinate
		centers_gpu[index].y=y_mean/count;
	}	

}


//calculating residual error(MSE) between previous and current centres
float error_calculation(point* centers_curr,point* centers_prev,int N)
{
  float err=0; //initialize MSE to zero
  for(int i=0;i<N;i++)
  {
    err+=pow((centers_curr[i].x-centers_prev[i].x),2) + pow((centers_curr[i].y-centers_prev[i].y),2); 
    //squared error between current and previous coordinates
  }

  err=((float)err)/N; //take mean of the squared error
  return err; //
}

int main(int argc, char* argv[])
{
	hipEvent_t start, stop, begin, end;//to store time intervals of execution

  	cout<<"Simple Linear Iterative Clustering: GPU IMPLEMENTATION"<<endl<<endl;

  	//create event, now these can be used for record 
	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));
	HANDLE_ERROR(hipEventCreate(&begin));
	HANDLE_ERROR(hipEventCreate(&end));

	HANDLE_ERROR(hipEventRecord(begin));//measure time for begin the read

	if(argc != 4) //there should be three arguments
	{
		cout<<" program_name image_name num_superpixels control_constant"<<endl;
		return 1; //exit and return an error
	}
	//READING FILE
	//reading file line by line

	ifstream infile;
	infile.open(argv[1]);  //opening the file
	string line;

	int img_wd, img_ht;
	int max_pixel_val;

	//line one contains P6, line 2 mentions about gimp version, line 3 stores the height and width
	getline(infile, line);
	istringstream iss1(line);

	//reading first line to check format
	int word;
	string str1;
	iss1>>str1;

	if(str1.compare("P6")!=0) //comparing magic number
	{
		cout<<"wrong file format"<<endl;
		return 1;
	}


	getline(infile,line); //this line has version related comment, hence ignoring
	getline(infile,line); //this stores image dims

	istringstream iss2(line);
	iss2>>word;// this will be image width
	img_wd=word;
	iss2>>word;// this will be image height
	img_ht=word;

	// cout<<img_ht<<" "<<img_wd<<endl;

	//storing the pixels as 1d images(row major)
	pixel_RGB *Pixel = (pixel_RGB*)malloc((img_ht)*(img_wd)*sizeof(pixel_RGB));

	int pix_cnt=0, cnt=0;

	getline(infile,line); //this stores max value
	istringstream iss3(line);
	iss3>>word;

	max_pixel_val=word;//max pixel value
	// cout<<max_pixel_val<<endl;
	unsigned int val;

	//read line by line
  	while (getline(infile, line))
  	{
    	istringstream iss4(line);
    	for (int i=0; i<=line.length();i++)
    	{
      		if(pix_cnt<img_ht*img_wd) //if it a valid pixel
		    {
		        val =((int)line[i]);  //read the current line
		        if(cnt%3==0)  //in case of R channel
		        {
		          Pixel[pix_cnt].r=val; //store R channel value
		        }
		        else if(cnt%3==1) //in case of G channel
		        {
		          Pixel[pix_cnt].g=val;//storing G value
		        }
		        else  
		        {
		          Pixel[pix_cnt].b=val;//in case of B channel, store it
		          pix_cnt++;  //move to next pixel
		        }
		        cnt++;  //next value read
		    }
    	}
  	}

	HANDLE_ERROR(hipEventRecord(stop));
	HANDLE_ERROR(hipEventSynchronize(stop));

	float milliseconds=0;
	HANDLE_ERROR(hipEventElapsedTime(&milliseconds, begin, stop));//time taken to read and save the image
	//get the time in milliseconds


	cout<<"Image read in "<<milliseconds<<" ms"<<endl;

	//COLOR CONVERSION
	//RGB->CIE-L*ab

	hipDeviceProp_t prop;
	HANDLE_ERROR(hipGetDeviceProperties(&prop,0));

	float thread_block=sqrt(prop.maxThreadsPerBlock);	//2D blocks used
	dim3 DimGrid(ceil(img_wd/thread_block),ceil(img_ht/thread_block),1); //image saved as a 2D grid
	dim3 DimBlock(thread_block,thread_block,1); //blocks are 2D

	pixel_RGB* Pixel_gpu; //to copy img to gpu
	HANDLE_ERROR(hipMalloc(&Pixel_gpu,img_ht*img_wd*sizeof(pixel_RGB)));//allocating memory on gpu for this
	HANDLE_ERROR(hipMemcpy(Pixel_gpu,Pixel,img_wd*img_ht*sizeof(pixel_RGB),hipMemcpyHostToDevice));//copying the rgb image value to gpu

	pixel_XYZ* Pixel_lab_gpu; //to store LAB image on gpu
	pixel_XYZ* Pixel_LAB=(pixel_XYZ*)malloc(img_ht*img_wd*sizeof(pixel_XYZ)); // to store LAB image on cpu

	HANDLE_ERROR(hipMalloc(&Pixel_lab_gpu,img_ht*img_wd*sizeof(pixel_XYZ)));//allocating LAB image memory on gpu
	HANDLE_ERROR(hipMemcpy(Pixel_lab_gpu,Pixel_LAB,img_wd*img_ht*sizeof(pixel_XYZ),hipMemcpyHostToDevice));//copying lab image values to gpu, currently random values

	HANDLE_ERROR(hipEventRecord(start)); //start recording kernel time
	// pixel_XYZ *Pixel_XYZ=RGB_XYZ(Pixel, img_ht, img_wd);
	RGB2LAB<<<DimGrid,DimBlock>>>(Pixel_gpu, img_wd, img_ht, Pixel_lab_gpu); //calling the kernel
	
	HANDLE_ERROR(hipEventRecord(stop));//stop recording kernel time
	HANDLE_ERROR(hipEventSynchronize(stop));//synchronizing 

	HANDLE_ERROR(hipMemcpy(Pixel_LAB,Pixel_lab_gpu,img_wd*img_ht*sizeof(pixel_XYZ),hipMemcpyDeviceToHost));//copying back the LAB values
	HANDLE_ERROR(hipFree(Pixel_lab_gpu));//frreing the cuda memory

	HANDLE_ERROR(hipEventElapsedTime(&milliseconds, start, stop));//get the time in milliseconds

	cout<<"Colorspace conversion done in "<<milliseconds<<" ms"<<endl;
	//IMPLEMENTING SLIC ALGORITHM
	int N = img_ht*img_wd;  //number of pixels in the images
	int K = atoi(argv[2]);    //number of superpixels desired

	int S= floor(sqrt(N/K));//size of each superpixel
	int m=atoi(argv[3]);    //compactness control constant
	int k1=ceil(img_ht*1.0/S)*ceil(img_wd*1.0/S);//actual number of superpixels

	cout<<"Image size: "<<img_wd<<" x "<<img_ht<<endl;
	cout<<"Using SLIC algorithm to get "<<k1<<" superpixels of approximate size "<<S<<" x "<<S<<", area "<<S*S<<" each, also m/S="<<1.0*m/S<<endl;
 
 	point* centers_curr=(point*)malloc(k1*sizeof(point));

	//initialize centers


	int center_ctr=0;
	//centres are initialized in a regular grid, each separated by S distance to the nearest centre
  	// centres start from (S/2,S/2)
	HANDLE_ERROR(hipEventRecord(start));

	for(int j=S/2;j<S*ceil(img_ht*1.0/S);j=j+S)
	{
		for(int i=S/2;i<S*ceil(img_wd*1.0/S);i=i+S)
		{
			int val1=((i>=img_wd)?(img_wd+j-S)/2:i);//to make sure it doesn't go out of image
			int val2=((j>=img_ht)?(img_ht+i-S)/2:j);//same as above in y coordinate
     	    //store x and y coordinates into the array
			centers_curr[center_ctr].x=val1;
			centers_curr[center_ctr].y=val2;
			center_ctr++;

		}
	}

	HANDLE_ERROR(hipEventRecord(stop));
	HANDLE_ERROR(hipEventSynchronize(stop));

	HANDLE_ERROR(hipEventElapsedTime(&milliseconds, start, stop));//get the time in milliseconds
	cout<<"centres initialized in "<<milliseconds<<" ms"<<endl;
	
	////perturb centers

	HANDLE_ERROR(hipEventRecord(start));
	float* G=(float*)malloc(N*sizeof(float)); 
	//to store gradient in 3x3 neighborhood
 	//gradient is calculated as : G(x, y) = \I(x + 1, y) − I(x − 1, y)|^2+ |I(x, y + 1) − I(x, y − 1)|^2
	
	for(int i=0; i<img_wd;i++)//x-coordinate
	{
	  for(int j=0; j<img_ht;j++)//y-coordinate
	  {
	    int index=j*img_wd+i;//calculating the index, row major

      //To store L,a, b channels for points (x+1,y),(x-1,y),(x,y+1),(x,y-1)
	    float L1, L2, L3, L4, a1, a2, a3, a4, b1, b2, b3, b4;

	    //initializing them to zero, so as to give padding effect when at edges

	    L1=L2=L3=L4=a1=a2=a3=a4=b1=b2=b3=b4=0;

		// pt1 is point(x+1, y),pt 2 is point(x-1,y),pt3 is point(x,y+1), pt4 is point(x,y-1)
	    //replace by actual intensities in LAB colorspace when the pixel exists

	    if(i+1<img_wd)
	      L1=Pixel_LAB[j*img_wd+i+1].x, a1=Pixel_LAB[j*img_wd+i+1].y, b1=Pixel_LAB[j*img_wd+i+1].z;
	    if(i-1>0)
	      L2=Pixel_LAB[j*img_wd+i-1].x, a2=Pixel_LAB[j*img_wd+i-1].y, b2=Pixel_LAB[j*img_wd+i-1].z;
	    if(j+1<img_ht)
	      L3=Pixel_LAB[(j+1)*img_wd+i].x, a3=Pixel_LAB[(j+1)*img_wd+i].y, b3=Pixel_LAB[(j+1)*img_wd+i].z;
	    if(j-1>0)
	      L4=Pixel_LAB[(j-1)*img_wd+i].x, a4=Pixel_LAB[(j-1)*img_wd+i].y, b4=Pixel_LAB[(j-1)*img_wd+i].z;
      	
      	//Calculating the gradient
	    G[index]=pow(L1-L2,2) + pow(a1-a2,2) + pow(b1-b2,2) + pow(L3-L4,2) + pow(a3-a4,2) + pow(b3-b4,2);
	  }
	}

	for(int i=0; i<k1;i++)  ////for every cluster center
	{
		//the minimum gradient is needed in the region (x-1,y-1) to (x+1,y+1)
		int x1=centers_curr[i].x-1;
		int x2=centers_curr[i].x+1;
	    int y1=centers_curr[i].y-1;
    	int y2=centers_curr[i].y+1;

		int index = min_index(G, N, x1, x2, y1, y2, img_wd);//finding minimum index in this 3x3 search region
	
	    //calculating new x and y coordinates for the centre 
		centers_curr[i].x=(floor)(index%img_wd);
		centers_curr[i].y=(floor)(index/img_wd);

	} 	

	HANDLE_ERROR(hipEventRecord(stop));
	HANDLE_ERROR(hipEventSynchronize(stop));

	HANDLE_ERROR(hipEventElapsedTime(&milliseconds, start, stop));//get the time in milliseconds
	cout<<"Centres perturbed in "<<milliseconds<<" ms"<<endl;
	
	HANDLE_ERROR(hipEventRecord(start));

	int* labels=(int*)malloc(N*sizeof(int));//this will be storing labels for every pixel
	float* d=(float*)malloc(N*sizeof(float)); // this will be storing distance measure of every pixel to its cluster center
	
	//initializing the labels and distance measures
	for(int idx=0;idx<N;idx++)
	{
	  labels[idx]=-1; //unlabelled 
	  d[idx]=60000;   //a high value 
	}
	HANDLE_ERROR(hipEventRecord(stop));
	HANDLE_ERROR(hipEventSynchronize(stop));

	HANDLE_ERROR(hipEventElapsedTime(&milliseconds, start, stop));//get the time in milliseconds
	cout<<"labels and distance measures initialized in "<<milliseconds<<" ms"<<endl;
	
	float error=100;// initialize error to a high value
	//label assignment

	point* centers_gpu; //for storing the centers in gpu
	float* d_gpu;		//for storing distance measures in gpu
	int* labels_gpu;	//for storing labels in gpu
	pixel_XYZ* Pixel_LAB_gpu;//for storing LAB image in gpu
	
	point* centers_prev=(point*)malloc(k1*sizeof(point));// this will be storing the cluster centres for every previous epoch

	HANDLE_ERROR(hipMalloc(&centers_gpu, k1*sizeof(point)));//allocating memory for centers on gpu

	HANDLE_ERROR(hipMalloc(&labels_gpu, N*sizeof(int)));//allocating memory for labels on gpu

	HANDLE_ERROR(hipMalloc(&Pixel_LAB_gpu, N*sizeof(pixel_XYZ)));//allocating memory for LAB image on gpu

	HANDLE_ERROR(hipMalloc(&d_gpu, N*sizeof(float)));//allocating memory for distance measures on gpu

	HANDLE_ERROR(hipMemcpy(Pixel_LAB_gpu, Pixel_LAB, N*sizeof(pixel_XYZ), hipMemcpyHostToDevice));//copying LAB_image from host to device
	unsigned int thread_block1=prop.maxThreadsPerBlock;//1D grid and block
	
	int epoch=0; //initialize epoch

	while(error>1)
	{
		cout<<endl<<"Epoch = "<<epoch<<endl;

		for(int i=0; i<k1;i++)//for every cluster centre
		{
	    	centers_prev[i].x=centers_curr[i].x; //find x coordinate of the cluster centre
			centers_prev[i].y=centers_curr[i].y; //find y coordinate of the cluster centre
		}

		HANDLE_ERROR(hipMemcpy(labels_gpu, labels, N*sizeof(int), hipMemcpyHostToDevice));//copying labels on gpu
		HANDLE_ERROR(hipMemcpy(centers_gpu, centers_curr, k1*sizeof(point), hipMemcpyHostToDevice));//copying centers on gpu

		HANDLE_ERROR(hipMemcpy(d_gpu, d , N*sizeof(float), hipMemcpyHostToDevice));//copying distance measures on gpu
			
		dim3 DimGrid1(1+(k1/thread_block1),1,1); //1D grid
		dim3 DimBlock1(thread_block1,1,1);//1D block

		HANDLE_ERROR(hipEventRecord(start));
		label_assignment<<<DimGrid1,DimBlock1>>>(labels_gpu,Pixel_LAB_gpu,centers_gpu,S,img_wd, img_ht,m, d_gpu, k1);
		HANDLE_ERROR(hipEventRecord(stop));
		HANDLE_ERROR(hipEventSynchronize(stop));

		HANDLE_ERROR(hipEventElapsedTime(&milliseconds, start, stop));//get the time in milliseconds
		cout<<"Label and distance assignment done in "<<milliseconds<<" ms"<<endl;
	
		HANDLE_ERROR(hipEventRecord(start));
		update_centres<<<DimGrid1,DimBlock1>>>(labels_gpu, centers_gpu, S, img_wd, img_ht, k1);
		HANDLE_ERROR(hipEventRecord(stop));
		HANDLE_ERROR(hipEventSynchronize(stop));

		HANDLE_ERROR(hipEventElapsedTime(&milliseconds, start, stop));//get the time in milliseconds
		cout<<"Centers updated in "<<milliseconds<<" ms"<<endl;
	
		HANDLE_ERROR(hipMemcpy(centers_curr, centers_gpu, k1*sizeof(point), hipMemcpyDeviceToHost)); //copying centers back to cpu
		HANDLE_ERROR(hipMemcpy(d, d_gpu, N*sizeof(float), hipMemcpyDeviceToHost));//copying distances back to cpu
		HANDLE_ERROR(hipMemcpy(labels, labels_gpu, N*sizeof(int), hipMemcpyDeviceToHost));//copying labels back to cpu

		HANDLE_ERROR(hipEventRecord(start));
		error= error_calculation(centers_curr, centers_prev,k1);
		HANDLE_ERROR(hipEventRecord(stop));
		HANDLE_ERROR(hipEventSynchronize(stop));

		HANDLE_ERROR(hipEventElapsedTime(&milliseconds, start, stop));//get the time in milliseconds
		
		cout<<"MSE = "<<error<<" and is calculated in "<<milliseconds<<" ms"<<endl;
		epoch++; //next epoch
	}


	//OUTPUT STORAGE


	pixel_RGB *rgb=(pixel_RGB*)malloc((img_ht)*(img_wd)*sizeof(pixel_RGB));
	HANDLE_ERROR(hipEventRecord(start));
	///enforce connectivity
	//for every point, look into its 4 neighbour labels, if all are same and different from pixel's label, change its label
	for(int x=0; x<img_wd; x++)
	{
	  for(int y=0; y<img_ht; y++)
	  {
	    //int L_0=labels[y*img_wd+x];
	    int L_t=labels[max(y-1,0)*img_wd+x];
	    int L_b=labels[min(y+1,img_ht)*img_wd+x];
	    int L_r=labels[y*img_wd+max(img_wd,x+1)];
	    int L_l=labels[y*img_wd+min(0,x-1)];

	    if(L_t==L_b && L_b==L_r && L_r==L_l)
	    {
	      labels[y*img_wd+x]=L_t;
	// cout<<"stray pixel found"<<endl;
	    }
	  }
	}

	HANDLE_ERROR(hipEventRecord(stop));
	HANDLE_ERROR(hipEventSynchronize(stop));

	HANDLE_ERROR(hipEventElapsedTime(&milliseconds, start, stop));//get the time in milliseconds
		

	cout<<"connectivity enforced in "<<milliseconds<<" ms"<<endl;

	//randomly shuffle the labels
	random_shuffle(labels,labels+k1);

	HANDLE_ERROR(hipEventRecord(start));

	float alpha=0;
	for(int i=0;i<img_ht*img_wd;i++)
	{
	  int label_val=labels[i];
	// cout<<label_val<<endl;
	  rgb[i].r=alpha*(21*label_val%255) + (1-alpha)*Pixel[i].r;
	  rgb[i].g=alpha*(47*label_val%255) + (1-alpha)*Pixel[i].g;
	  rgb[i].b=alpha*(173*label_val%255) + (1-alpha)*Pixel[i].b;
	}

	//sobel edge detection
	int valX, valY = 0; 
	int GX [3][3];
	int GY [3][3];

	//Sobel Horizontal Mask     
	GX[0][0] = 1; GX[0][1] = 0; GX[0][2] = -1; 
	GX[1][0] = 2; GX[1][1] = 0; GX[1][2] = -2;  
	GX[2][0] = 1; GX[2][1] = 0; GX[2][2] = -1;

	//Sobel Vertical Mask   
	GY[0][0] =  1; GY[0][1] = 2; GY[0][2] =   1;    
	GY[1][0] =  0; GY[1][1] = 0; GY[1][2] =   0;    
	GY[2][0] = -1; GY[2][1] =-2; GY[2][2] =  -1;

	double val1;
	for(int i=0;i<img_wd;i++)
	{
		for(int j=0;j<img_ht;j++)
		{
			if(i==0||i==img_wd-1||j==0||j==img_ht-1)
			{
				valX=0;
				valY=0;
			}
			else
			{
				valX=0, valY=0;
				for (int x = -1; x <= 1; x++)
				{
					for (int y = -1; y <= 1; y++)
					{
					  valX = valX + labels[i+x+(j+y)*img_wd] * GX[1+x][1+y];
					  valY = valY + labels[i+x+(j+y)*img_wd]  * GY[1+x][1+y];
					}
				}

			}
			val1=sqrt(valX*valX + valY*valY);
			if(val1>0)
			{
				rgb[j*img_wd+i].r=0;
				rgb[j*img_wd+i].g=0;
				rgb[j*img_wd+i].b=0;
			}
		}
	}
	  
	HANDLE_ERROR(hipEventRecord(stop));
	HANDLE_ERROR(hipEventSynchronize(stop));

	HANDLE_ERROR(hipEventElapsedTime(&milliseconds, start, stop));//get the time in milliseconds
		

	cout<<"Output image prepared in "<<milliseconds <<" ms"<<endl;


	// //OUTPUT STORAGE
	HANDLE_ERROR(hipEventRecord(start));
	ofstream ofs;
	ofs.open("output_gpu.ppm", ofstream::out);
	ofs<<"P6\n"<<img_wd<<" "<<img_ht<<"\n"<<max_pixel_val<<"\n";

	for(int j=0; j <img_ht*img_wd;j++)
	{
		ofs<<rgb[j].r<<rgb[j].g<<rgb[j].b;//labelled_ini[j]<<0<<0;//ofs<<Pixel_LAB[j].x<<Pixel_LAB[j].y<<Pixel_LAB[j].z; //write as ascii
	}

	ofs.close();
	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&milliseconds, start, end);//get the time in milliseconds

	cout<<"Image saved in "<<milliseconds<<" ms"<<endl;
	hipEventElapsedTime(&milliseconds, begin, end);//get time for whole clustering

	cout<<"Clustering done in "<<milliseconds<<" ms"<<endl;

	return 0;
}