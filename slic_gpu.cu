	  #include <fstream>
	  #include <iostream>
	  #include <stdio.h>
	  #include <string>
	  #include <sstream>
	  #include <stdlib.h>
	  #include <math.h>
	  #include <time.h>
	  #include <ctime>
	  #include <vector>
	  #include <cstdlib>
	  #include <algorithm>
	  #include <hip/hip_runtime_api.h>
	  #include <hip/hip_runtime.h>


using namespace std;

	  //handlerror declaration : to display file and line numbers of erroneous lines
static void HandleError( hipError_t err, const char *file, int line ) {
	if (err != hipSuccess) {
		cout<<hipGetErrorString(err)<<" in "<< file <<" at line "<< line<<endl;
	}
}

	  #define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

	  // storing RGB values for rgb colorspace images
struct pixel_RGB
{
	    unsigned char r;  //Red values
	    unsigned char g;  //Green values
	    unsigned char b;  //Blue Values
	  };

	  // storing values for xyz and lab colorspace images
	  struct pixel_XYZ
	  {
	    double x;  //X for XYZ colorspace, L for LAB colorspace
	    double y;  //Y for XYZ colorspace, A for LAB colorspace
	    double z;  //Z for XYZ colorspace, B for LAB colorspace
	  };

	  //store coordinates for each cluster centres
	  struct point
	  { 
	    int x;  //x-ccordinate
	    int y;  //y-coordinate
	  };



	  //color space conversion from RGB to XYZ
	  pixel_XYZ* RGB_LAB(pixel_RGB* img ,int ht ,int wd)
	  { 
	    pixel_XYZ *LAB_img=(pixel_XYZ*)(malloc(ht*wd*sizeof(pixel_XYZ))); //declaring same sized output image

	    for(int i=0; i<ht*wd;i++)
	    {
	    	int R=img[i].r;
	    	int G=img[i].g;
	    	int B=img[i].b;

	    	double var_R=double(R)/255;
	    	double var_G=double(G)/255;
	    	double var_B=double(B)/255;

	    	double X = var_R * 0.4124 + var_G * 0.3576 + var_B * 0.1805;
	    	double Y = var_R * 0.2126 + var_G * 0.7152 + var_B * 0.0722;
	    	double Z = var_R * 0.0193 + var_G * 0.1192 + var_B * 0.9505;

	    	X=X/0.95047;
	  		Y=Y/1.00000;
	  		Z=Z/1.088969;

	  		double Y3=pow(Y,1/3);

	  		double T=0.008856;
	  		double fx=(X>T)?(pow(X,double(1)/3)):(7.787*X+(16/116));
	  		double fy=(Y>T)?(pow(Y,double(1)/3)):(7.787*Y+(16/116));
	  		double fz=(Z>T)?(pow(Z,double(1)/3)):(7.787*Z+(16/116));


	  		double L=(Y>T)?(116*Y3 - 16):(903.3*Y);
	      double a = 500 * (fx - fy);
	      double b = 200 * (fy - fz);

	      LAB_img[i].x=L;
	      LAB_img[i].y=a;
	      LAB_img[i].z=b;
	    }

	    return LAB_img;
	  }
	

	  int min_index(double* array, int size, int x1, int x2, int y1, int y2, int img_wd) //find the index of min value a given region
	  {
	  	int index=0;
	  	for(int i=0;i<size;i++)
	  	{
	      if(int(i%img_wd)>=x1 && int(i%img_wd)<=x2 && int(i/img_wd)>=y1 && int(i/img_wd)<=y2)//check if it is in the region of search
	      { 
	      	if(array[i]<array[index])
	      		index=i;
	      }
	    }
	    return index;
	  }


	  __global__ void label_assignment(int* labels_gpu, pixel_XYZ* Pixel_LAB_gpu, point* centers_gpu, int S, int img_wd, int img_ht,
	  	int m, double* d_gpu, int k1)
	  {
	  size_t index = blockIdx.x*blockDim.x+ threadIdx.x; //find threadindex of cluster center
		// finding centre coordinates
	  int x_center=centers_gpu[index].x;
	  int y_center=centers_gpu[index].y;
	  int centre_idx=y_center*img_wd+x_center;//find index in image row major form
	  labels_gpu[centre_idx]=index;
	  if(index>=k1) //for degenerate cases
	  	return;	

		for(int x_coord=max(0,x_center-S);x_coord<=min(img_wd,x_center+S);x_coord++) //look in 2S x 2S neighborhood
		{
			for(int y_coord=max(0,y_center-S);y_coord<=min(img_ht,y_center+S);y_coord++)
			{
				int j=y_coord*img_wd+x_coord; // find global index of the pixel
			  double d_c = powf(powf((Pixel_LAB_gpu[centre_idx].x-Pixel_LAB_gpu[j].x),2) + powf((Pixel_LAB_gpu[centre_idx].y-Pixel_LAB_gpu[j].y),2) + powf((Pixel_LAB_gpu[centre_idx].z-Pixel_LAB_gpu[j].z),2),0.5); //color proximity;
	   		double d_s = powf(powf(x_coord-x_center,2)+powf(y_coord-y_center,2),0.5); //spatial proximity
	   		double D=powf(powf(d_c,2)+powf(m*d_s/S,2),0.5);
	   		// printf("%d, %d ,%0.12lf, %0.12lf \n ",j,index,d_gpu[j],D);

	   		if(D<d_gpu[j])
	   		{
	   			d_gpu[j]=D;
	   			labels_gpu[j]=index;
	   		}
	   	}
	   }
	 }

	 __global__ void update_centres(int* labels_gpu, point* centers_gpu, int S, int img_wd, int img_ht, int k1)
	 {
		 size_t index = blockIdx.x*blockDim.x+ threadIdx.x; //thread index
		 // printf("index: %d \n",index);
		 if(index>=k1)
		 	return;
		// finding centre coordinates
		 int centre_x=centers_gpu[index].x;
		 int centre_y=centers_gpu[index].y;
	  //finding the label of cluster, this will be center's label
	  int i=labels_gpu[centre_y*img_wd+centre_x]; //finding the label of centre

	  int x_mean=0, y_mean=0, count=0, flag=0;
	  for(int x_coord=max(0,centre_x-S);x_coord<=min(img_wd,centre_x+S);x_coord++)
	  {
	  	for(int y_coord=max(0,centre_y-S);y_coord<=min(img_ht,centre_y+S);y_coord++)
	  	{
	  		int pt_idx=y_coord*img_wd+x_coord;

	  		if(labels_gpu[pt_idx]==i)
	  		{
	  			x_mean+=x_coord; 
	  			y_mean+=y_coord;
	  			flag++;
	  			count++;
	  		}
	  	}
	  }
	  if(flag)
	  {
	  	centers_gpu[index].x=x_mean/count;
	  	centers_gpu[index].y=y_mean/count;
	  }	
	  // printf("index: %d, initial values : %d %d , new values : %d %d \n",index, centre_x,centre_y,centers_gpu[index].x,centers_gpu[index].y);

	}

	

	
	

	double error_calculation(point* centers_curr,point* centers_prev,int N)
	{
		double err=0;
		for(int i=0;i<N;i++)
		{
			err+=pow((centers_curr[i].x-centers_prev[i].x),2) + pow((centers_curr[i].y-centers_prev[i].y),2);
	    // cout<<i<<" "<<"curr = ("<<centers_curr[i].x<<","<<centers_curr[i].y<<") , prev= ("<<centers_prev[i].x<<","<<centers_prev[i].y<<")"<<endl;
		}

		err=pow(((double)err),0.5)/N;
		return err;
	}

	int main(int argc, char* argv[])
	{
	    // time_t start=time(NULL);
		hipEvent_t start, stop;

		hipEventCreate(&start);
		hipEventCreate(&stop);
	    if(argc != 4) //there should be three arguments
	    {
	    	cout<<" program_name image_name num_superpixels control_constant"<<endl;
	      return 1; //exit and return an error
	    }
	    //READING FILE
	    
	    ifstream infile;
	    infile.open(argv[1]);  //opening the file
	    string line;

	    int img_wd, img_ht;
	    int max_pixel_val;

	    //line one contains P6, line 2 mentions about gimp version, line 3 stores the height and width
	    getline(infile, line);
	    istringstream iss1(line);

	    //reading first line to check format
	    int word;
	    string str1;
	    iss1>>str1;
	    
	    if(str1.compare("P6")!=0) //comparing magic number
	    {
	    	cout<<"wrong file format"<<endl;
	    	return 1;
	    }
	    
	    hipEventRecord(start);

	    getline(infile,line); //this line has version related comment, hence ignoring
	    getline(infile,line); //this stores image dims

	    istringstream iss2(line);
	    iss2>>word;// this will be image width
	    img_wd=word;
	    iss2>>word;// this will be image height
	    img_ht=word;
	    
	    // cout<<img_ht<<" "<<img_wd<<endl;

	    //storing the pixels as 1d images
	    pixel_RGB *Pixel = (pixel_RGB*)malloc((img_ht)*(img_wd)*sizeof(pixel_RGB));
	    
	    int pix_cnt=0, cnt=0;

	    getline(infile,line); //this stores max value
	    istringstream iss3(line);
	    iss3>>word;

	    max_pixel_val=word;//max pixel value
	    // cout<<max_pixel_val<<endl;
	    unsigned int val;

	    while (getline(infile, line))
	    {
	    	istringstream iss4(line);
	    	for (int i=0; i<=line.length();i++)
	    	{
	    		if(pix_cnt<img_ht*img_wd)
	    		{
	    			val =((int)line[i]);
	    			if(cnt%3==0)
	    			{
	    				Pixel[pix_cnt].r=val;
	    			}
	    			else if(cnt%3==1)
	    			{
	    				Pixel[pix_cnt].g=val;
	    			}
	    			else
	    			{
	    				Pixel[pix_cnt].b=val;
	    				pix_cnt++;
	    			}
	    			cnt++;
	    		}
	    	}
	    }
	    hipEventRecord(stop);
	    hipEventSynchronize(stop);

	    float milliseconds=0;
			hipEventElapsedTime(&milliseconds, start, stop);//get the time in milliseconds


			cout<<"File read in "<<milliseconds<<" ms"<<endl;

	    //COLOR CONVERSION
	    //RGB->XYZ->CIE-L*ab

	    //RGB to XYZ
	    // time_t t9= time(NULL);
			hipEventRecord(start);
			// pixel_XYZ *Pixel_XYZ=RGB_XYZ(Pixel, img_ht, img_wd);
			
	    //XYZ TO CIE-L*ab
			pixel_XYZ* Pixel_LAB=RGB_LAB(Pixel, img_ht, img_wd);
			hipEventRecord(stop);
			hipEventSynchronize(stop);

	    hipEventElapsedTime(&milliseconds, start, stop);//get the time in milliseconds

	    cout<<"Colorspace conversion done in "<<milliseconds<<" ms"<<endl;
	    //IMPLEMENTING SLIC ALGORITHM
	    int N = img_ht*img_wd;  //number of pixels in the images
	    int K = atoi(argv[2]);    //number of superpixels desired

	    int S= floor(sqrt(N/K));//size of each superpixel
	    int m=atoi(argv[3]);    //compactness control constant
	    int k1=ceil(img_ht*1.0/S)*ceil(img_wd*1.0/S);//actual number of superpixels

	    cout<<"Image size: "<<img_wd<<" x "<<img_ht<<endl;
	    cout<<"Using SLIC algorithm to get "<<k1<<" superpixels of approximate size "<<S<<" x "<<S<<", area "<<S*S<<" each, also m/S="<<1.0*m/S<<endl;
	    // cout<<k1<<" "<<S<<" "<<float(img_ht*1.0/S)<<" "<<float(img_wd*1.0/S)<<endl;
	    point* centers_curr=(point*)malloc(k1*sizeof(point));

	    //initialize centers

	    hipEventRecord(start);

	    int center_ctr=0;
	    for(int j=S/2;j<S*ceil(img_ht*1.0/S);j=j+S)
	    {
	    	for(int i=S/2;i<S*ceil(img_wd*1.0/S);i=i+S)
	    	{
	    		int val1=((i>=img_wd)?(img_wd+j-S)/2:i);
	    		int val2=((j>=img_ht)?(img_ht+i-S)/2:j);
	    		centers_curr[center_ctr].x=val1;
	    		centers_curr[center_ctr].y=val2;
	        // cout<<center_ctr<<" "<<centers_curr[center_ctr].x<<" "<<centers_curr[center_ctr].y<<" "<<val1<<" "<<val2<<endl;
	    		center_ctr++;

	    	}
	    }
	    
	    hipEventRecord(stop);
	    hipEventSynchronize(stop);

	    hipEventElapsedTime(&milliseconds, start, stop);//get the time in milliseconds
	    cout<<"centres initialized in "<<milliseconds<<" ms"<<endl;
	    //perturb centers

	    hipEventRecord(start);
	    double* G=(double*)malloc(N*sizeof(double)); 
	    for(int i=0; i<img_wd;i++)
	    {
	    	for(int j=0; j<img_ht;j++)
	    	{
	    		int index=j*img_wd+i;
	    		double L1, L2, L3, L4, a1, a2, a3, a4, b1, b2, b3, b4;
	    		L1=L2=L3=L4=a1=a2=a3=a4=b1=b2=b3=b4=0;

	     		// cout<<i<<" "<<j<<endl;
	     		// pt1 is point(x+1, y),pt 2 is point(x-1,y),pt3 is point(x,y+1), pt4 is point(x,y-1)

	    		if(i+1<img_wd)
	    			L1=Pixel_LAB[j*img_wd+i+1].x, a1=Pixel_LAB[j*img_wd+i+1].y, b1=Pixel_LAB[j*img_wd+i+1].z;
	    		if(i-1>0)
	    			L2=Pixel_LAB[j*img_wd+i-1].x, a2=Pixel_LAB[j*img_wd+i-1].y, b2=Pixel_LAB[j*img_wd+i-1].z;
	    		if(j+1<img_ht)
	    			L3=Pixel_LAB[(j+1)*img_wd+i].x, a3=Pixel_LAB[(j+1)*img_wd+i].y, b3=Pixel_LAB[(j+1)*img_wd+i].z;
	    		if(j-1>0)
	    			L4=Pixel_LAB[(j-1)*img_wd+i].x, a4=Pixel_LAB[(j-1)*img_wd+i].y, b4=Pixel_LAB[(j-1)*img_wd+i].z;

	    		G[index]=pow(L1-L2,2) + pow(a1-a2,2) + pow(b1-b2,2) + pow(L3-L4,2) + pow(a3-a4,2) + pow(b3-b4,2);
	    	}
	    }
	    
	    hipDeviceProp_t prop;

	    for(int i=0; i<k1;i++)  //for every component
	    {
	    	int x1=centers_curr[i].x-1;
	    	int x2=centers_curr[i].x+1;
	    	int y1=centers_curr[i].y-1;
	    	int y2=centers_curr[i].y+1;

	    	int index = min_index(G, N, x1, x2, y1, y2, img_wd);
	    	centers_curr[i].x=(floor)(index%img_wd);
	    	centers_curr[i].y=(floor)(index/img_wd);

	    } 
	    hipEventRecord(stop);
	    hipEventSynchronize(stop);

	    hipEventElapsedTime(&milliseconds, start, stop);//get the time in milliseconds
	    cout<<"centres perturbed in "<<milliseconds<<" ms"<<endl;
	    
	    ///label initialized to all -1
	    int* labels=(int*)malloc(N*sizeof(int));
	    double* d=(double*)malloc(N*sizeof(double));
	    
	    hipEventRecord(start);	    
	    for(int idx=0;idx<N;idx++)
	    {
	    	labels[idx]=-1; 
	    	d[idx]=60000.00;
	    }
	    hipEventRecord(stop);
	    hipEventSynchronize(stop);

	    hipEventElapsedTime(&milliseconds, start, stop);//get the time in milliseconds
	    
	    cout<<"labels and distance measures initialized in "<<milliseconds<<" ms"<<endl;

	    double error=100;

	    point* centers_prev=(point*)(malloc(k1*sizeof(point)));
	    int epoch=0;
	    while(error>10)
	    {
	    	cout<<"Epoch number "<<epoch<<endl;


	    	point* centers_gpu;
	    	double* d_gpu;
	    	int* labels_gpu;
	    	pixel_XYZ* Pixel_LAB_gpu;
	    	HANDLE_ERROR(hipMalloc(&centers_gpu, k1*sizeof(point)));

	    	HANDLE_ERROR(hipMalloc(&labels_gpu, N*sizeof(int)));

	    	HANDLE_ERROR(hipMalloc(&Pixel_LAB_gpu, N*sizeof(pixel_XYZ)));
	    	HANDLE_ERROR(hipMalloc(&d_gpu, N*sizeof(double)));
	    	hipDeviceProp_t prop;

	    	unsigned int thread_block1=prop.maxThreadsPerBlock;
	    	// cout<<N<<" "<<S<<" "<<K<<" "<<k1<<" "<<thread_block1<<endl;

	    	dim3 DimGrid1(1+(k1/thread_block1),1,1); 
	    	dim3 DimBlock1(thread_block1,1,1);
	    	// cout<<DimGrid1.x<<" "<<DimBlock1.x<<endl;

	    	HANDLE_ERROR(hipMemcpy(labels_gpu, labels, N*sizeof(int), hipMemcpyHostToDevice));
	    	HANDLE_ERROR(hipMemcpy(centers_gpu, centers_curr, k1*sizeof(point), hipMemcpyHostToDevice));
	    	HANDLE_ERROR(hipMemcpy(Pixel_LAB_gpu, Pixel_LAB, N*sizeof(pixel_XYZ), hipMemcpyHostToDevice));
	    	HANDLE_ERROR(hipMemcpy(d_gpu, d , N*sizeof(double), hipMemcpyHostToDevice));
	    	// for(int i=0; i<N;i++)
	    		// cout<<labels[i]<<endl;
	    	hipEventRecord(start);

	    	label_assignment<<<DimGrid1,DimBlock1>>>(labels_gpu,Pixel_LAB_gpu,centers_gpu,S,img_wd, img_ht,m, d_gpu, k1);

	    	hipEventRecord(stop);
	    	hipEventSynchronize(stop);

	    hipEventElapsedTime(&milliseconds, start, stop);//get the time in milliseconds
	    

	    	cout<<"Distances calculated for all points neighboured to centres in "<<milliseconds<<" ms"<<endl;// in "<<double(t2-t1)<<" secs"<<endl;

	    //update cluster centres
	    	// t1=time(NULL);

	    	for(int i=0; i<k1;i++)
	    	{
	    centers_prev[i].x=centers_curr[i].x; //saving current centres, before any recalculation
	    centers_prev[i].y=centers_curr[i].y;
	  }
	  hipEventRecord(start);
	  update_centres<<<DimGrid1,DimBlock1>>>(labels_gpu, centers_gpu, S, img_wd, img_ht, k1);
	  hipEventRecord(stop);
	  hipEventSynchronize(stop);
	  hipEventSynchronize(stop);

	    hipEventElapsedTime(&milliseconds, start, stop);//get the time in milliseconds
	    
	//copy back centres, labels, d
	    HANDLE_ERROR(hipMemcpy(centers_curr, centers_gpu, k1*sizeof(point), hipMemcpyDeviceToHost));
	    HANDLE_ERROR(hipMemcpy(d, d_gpu, N*sizeof(int), hipMemcpyDeviceToHost));
	    HANDLE_ERROR(hipMemcpy(labels, labels_gpu, N*sizeof(int), hipMemcpyDeviceToHost));

	  // t2=time(NULL);
	    cout<<"cluster centers updated in "<<milliseconds<<" ms"<<endl;

	    //error calculation
	    hipEventRecord(start);
	    error= error_calculation(centers_curr, centers_prev,k1);
	    hipEventRecord(stop);
	    hipEventSynchronize(stop);

	    hipEventElapsedTime(&milliseconds, start, stop);//get the time in milliseconds
	    
	    cout<<"error = "<<error<<" and is calculated in "<<milliseconds<<" ms"<<endl;
	    HANDLE_ERROR(hipFree(labels_gpu));
	    HANDLE_ERROR(hipFree(Pixel_LAB_gpu));
	    HANDLE_ERROR(hipFree(centers_gpu));
	    HANDLE_ERROR(hipFree(d_gpu));
	    epoch++;
	  }


//enforcing connectivity
	  //for every pixel, find if it is stray, by analysising labels in all 4 directions
	  //if none is same as the pixel, change it to their
	  // for(int x=0; x<img_wd;x++)
	  // {
	  // 	for(int y=0; y<img_ht; y++)
	  // 	{
	  // 		//for the current pixel, get label
	  // 		int L_0=labels[y*img_wd+x];
	  // 		if(L_0!=labels[max(0,y-1)*img_wd+x] && L_0!=labels[min(y,img_ht)*img_wd+x] && L_0!=labels[y*img_wd+min(x+1,img_wd)] && L_0!=labels[y*img_wd+max(0,x-1)])	//comparing with top pixel
			// 	labels[y*img_wd+x]=labels[max(0,y-1)*img_wd+x];	
	  // 	}
	  // }


	  pixel_RGB *rgb=(pixel_RGB*)malloc((img_ht)*(img_wd)*sizeof(pixel_RGB));



	//randomly shuffle the labels
	  random_shuffle(labels,labels+k1);
	  float alpha=1;
	  hipEventRecord(start);
	  for(int i=0;i<img_ht*img_wd;i++)
	  {
	  	int label_val=labels[i];
	      // cout<<label_val<<endl;
	  	rgb[i].r=alpha*(21*label_val%255);// + (1-alpha)*Pixel[i].r;
	  	rgb[i].g=alpha*(47*label_val%255) ;//+ (1-alpha)*Pixel[i].g;
	  	rgb[i].b=alpha*(173*label_val%255) ;//+ (1-alpha)*Pixel[i].b;
	  }

	    //labelling the centers
	  for(int i=0; i<k1;i++)  
	  {
	  	int x_coord=centers_curr[i].x;
	  	int y_coord=centers_curr[i].y;
	      // cout<<x_coord<<" "<<y_coord<<endl;
	  	for (int x=x_coord-5; x<x_coord+5; x++)
	  	{
	  		for(int y=y_coord-5; y<y_coord+5; y++)
	  		{
	  			int idx=img_wd*y_coord + x_coord;
	          rgb[idx].r= 0;//NULL;//(unsigned char) 0; 
	          rgb[idx].g= 0;//(unsigned char) 0; 
	          rgb[idx].b= 0;//(unsigned char) 0;
	        }
	      }
	    }
	    hipEventRecord(stop);
	    hipEventSynchronize(stop);
	    hipEventElapsedTime(&milliseconds, start, stop);//get the time in milliseconds

	    cout<<"Image prepared in "<<milliseconds<<" ms"<<endl;

	    //OUTPUT STORAGE
	    hipEventRecord(start);
	    ofstream ofs;
	    ofs.open("output1.ppm", ofstream::out);
	    ofs<<"P6\n"<<img_wd<<" "<<img_ht<<"\n"<<max_pixel_val<<"\n";

	    for(int j=0; j <img_ht*img_wd;j++)
	    	ofs<<rgb[j].r<<rgb[j].g<<rgb[j].b;

	    ofs.close();
	    hipEventRecord(stop);
	    hipEventSynchronize(stop);
	    hipEventElapsedTime(&milliseconds, start, stop);//get the time in milliseconds

	    cout<<"Image saved in "<<milliseconds<<" ms"<<endl;
	    return 0;
	  }