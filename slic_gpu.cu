			  #include <fstream>
			  #include <iostream>
			  #include <stdio.h>
			  #include <string>
			  #include <sstream>
			  #include <stdlib.h>
			  #include <math.h>
			  #include <time.h>
			  #include <ctime>
			  #include <vector>
			  #include <cstdlib>
			  #include <algorithm>
			  #include <hip/hip_runtime_api.h>
			  #include <hip/hip_runtime.h>


using namespace std;

			  //handlerror declaration : to display file and line numbers of erroneous lines
static void HandleError( hipError_t err, const char *file, int line ) {
	if (err != hipSuccess) {
		cout<<hipGetErrorString(err)<<" in "<< file <<" at line "<< line<<endl;
	}
}

			  #define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

			  // storing RGB values for rgb colorspace images
struct pixel_RGB
{
			    unsigned char r;  //Red values
			    unsigned char g;  //Green values
			    unsigned char b;  //Blue Values
			};

			  // storing values for xyz and lab colorspace images
			struct pixel_XYZ
			{
			    double x;  //X for XYZ colorspace, L for LAB colorspace
			    double y;  //Y for XYZ colorspace, A for LAB colorspace
			    double z;  //Z for XYZ colorspace, B for LAB colorspace
			};

			  //store coordinates for each cluster centres
			struct point
			{ 
			    int x;  //x-ccordinate
			    int y;  //y-coordinate
			};



			  //color space conversion from RGB to LAB
			pixel_XYZ* RGB_LAB(pixel_RGB* img ,int ht ,int wd)
			{ 
			    pixel_XYZ *LAB_img=(pixel_XYZ*)(malloc(ht*wd*sizeof(pixel_XYZ))); //declaring same sized output image

			    for(int i=0; i<ht*wd;i++)
			    {
			    	int R=img[i].r;
			    	int G=img[i].g;
			    	int B=img[i].b;

			    	double var_R=double(R)/255;
			    	double var_G=double(G)/255;
			    	double var_B=double(B)/255;

			    	double X = var_R * 0.4124 + var_G * 0.3576 + var_B * 0.1805;
			    	double Y = var_R * 0.2126 + var_G * 0.7152 + var_B * 0.0722;
			    	double Z = var_R * 0.0193 + var_G * 0.1192 + var_B * 0.9505;

			    	X=X/0.95047;
			    	Y=Y/1.00000;
			    	Z=Z/1.088969;

			    	double Y3=pow(Y,1/3);

			    	double T=0.008856;
			    	double fx=(X>T)?(pow(X,double(1)/3)):(7.787*X+(16/116));
			    	double fy=(Y>T)?(pow(Y,double(1)/3)):(7.787*Y+(16/116));
			    	double fz=(Z>T)?(pow(Z,double(1)/3)):(7.787*Z+(16/116));


			    	double L=(Y>T)?(116*Y3 - 16):(903.3*Y);
			    	double a = 500 * (fx - fy);
			    	double b = 200 * (fy - fz);

			    	LAB_img[i].x=L;
			    	LAB_img[i].y=a;
			    	LAB_img[i].z=b;
			    }

			    return LAB_img;
			}
			

			  int min_index(double* array, int size, int x1, int x2, int y1, int y2, int img_wd) //find the index of min value a given region
			  {
			  	int index=0;
			  	for(int i=0;i<size;i++)
			  	{
			      if(int(i%img_wd)>=x1 && int(i%img_wd)<=x2 && int(i/img_wd)>=y1 && int(i/img_wd)<=y2)//check if it is in the region of search
			      { 
			      	if(array[i]<array[index])
			      		index=i;
			      }
			  }
			  return index;
			}


			__global__ void label_assignment(int* labels_gpu, pixel_XYZ* Pixel_LAB_gpu, point* centers_gpu, int S, int img_wd, int img_ht,
				int m, double* d_gpu, int k1)
			{
			  size_t index = blockIdx.x*blockDim.x+ threadIdx.x; //find threadindex of cluster center
			  
				// finding centre coordinates
			  int x_center=centers_gpu[index].x;
			  int y_center=centers_gpu[index].y;
			  int centre_idx=y_center*img_wd+x_center;//find index in image row major form
			  labels_gpu[centre_idx]=index;
			  if(index>=k1) //for degenerate cases
			  	return;	

				for(int x_coord=max(0,x_center-S);x_coord<=min(img_wd,x_center+S);x_coord++) //look in 2S x 2S neighborhood
				{
					for(int y_coord=max(0,y_center-S);y_coord<=min(img_ht,y_center+S);y_coord++)
					{
						int j=y_coord*img_wd+x_coord; // find global index of the pixel
					  	double d_c = powf(powf((Pixel_LAB_gpu[centre_idx].x-Pixel_LAB_gpu[j].x),2) + powf((Pixel_LAB_gpu[centre_idx].y-Pixel_LAB_gpu[j].y),2) + powf((Pixel_LAB_gpu[centre_idx].z-Pixel_LAB_gpu[j].z),2),0.5); //color proximity;
			   			double d_s = powf(powf(x_coord-x_center,2)+powf(y_coord-y_center,2),0.5); //spatial proximity
			   			double D=powf(powf(d_c,2)+powf(m*d_s/S,2),0.5);
			   			//printf("blockIdx=%d, blockDim=%d, threadidx=%d,m=%d, S=%d ,j=%d, index=%d ,d_gpu=%0.12lf, D=%0.12lf \n ",blockIdx.x,blockDim.x, threadIdx.x,m,S,j,index,d_gpu[j],D);
				   		 //printf("");
				   		if(D<d_gpu[j])
				   		{
				   			d_gpu[j]=D;
				   			labels_gpu[j]=index;
				   		}
				   	}
			   }
			}

			__global__ void update_centres(int* labels_gpu, point* centers_gpu, int S, int img_wd, int img_ht, int k1)
			{
				 size_t index = blockIdx.x*blockDim.x+ threadIdx.x; //thread index
				 // printf("index: %d \n",index);
				 if(index>=k1)
				 	return;
				// finding centre coordinates
				 int centre_x=centers_gpu[index].x;
				 int centre_y=centers_gpu[index].y;
			  //finding the label of cluster, this will be center's label
			  int i=labels_gpu[centre_y*img_wd+centre_x]; //finding the label of centre

			  int x_mean=0, y_mean=0, count=0, flag=0;
			  for(int x_coord=max(0,centre_x-S);x_coord<=min(img_wd,centre_x+S);x_coord++)
			  {
			  	for(int y_coord=max(0,centre_y-S);y_coord<=min(img_ht,centre_y+S);y_coord++)
			  	{
			  		int pt_idx=y_coord*img_wd+x_coord;

			  		if(labels_gpu[pt_idx]==i)
			  		{
			  			x_mean+=x_coord; 
			  			y_mean+=y_coord;
			  			flag++;
			  			count++;
			  		}
			  	}
			  }
			  if(flag)
			  {
			  	centers_gpu[index].x=x_mean/count;
			  	centers_gpu[index].y=y_mean/count;
			  }	
			//printf("index: %d, initial values : %d %d , new values : %d %d \n",index, centre_x,centre_y,centers_gpu[index].x,centers_gpu[index].y);

			}

			

			
			

			double error_calculation(point* centers_curr,point* centers_prev,int N)
			{
				double err=0;
				for(int i=0;i<N;i++)
				{
					err+=pow((centers_curr[i].x-centers_prev[i].x),2) + pow((centers_curr[i].y-centers_prev[i].y),2);
			    // cout<<i<<" "<<"curr = ("<<centers_curr[i].x<<","<<centers_curr[i].y<<") , prev= ("<<centers_prev[i].x<<","<<centers_prev[i].y<<")"<<endl;
				}

				err=pow(((double)err),0.5)/N;
				return err;
			}

			int main(int argc, char* argv[])
			{
			    // time_t start=time(NULL);
				hipEvent_t start, stop;

				hipEventCreate(&start);
				hipEventCreate(&stop);
			    if(argc != 4) //there should be three arguments
			    {
			    	cout<<" program_name image_name num_superpixels control_constant"<<endl;
			      return 1; //exit and return an error
			  }
			    //READING FILE

			  ifstream infile;
			    infile.open(argv[1]);  //opening the file
			    string line;

			    int img_wd, img_ht;
			    int max_pixel_val;

			    //line one contains P6, line 2 mentions about gimp version, line 3 stores the height and width
			    getline(infile, line);
			    istringstream iss1(line);

			    //reading first line to check format
			    int word;
			    string str1;
			    iss1>>str1;
			    
			    if(str1.compare("P6")!=0) //comparing magic number
			    {
			    	cout<<"wrong file format"<<endl;
			    	return 1;
			    }
			    
			    hipEventRecord(start);

			    getline(infile,line); //this line has version related comment, hence ignoring
			    getline(infile,line); //this stores image dims

			    istringstream iss2(line);
			    iss2>>word;// this will be image width
			    img_wd=word;
			    iss2>>word;// this will be image height
			    img_ht=word;
			    
			    // cout<<img_ht<<" "<<img_wd<<endl;

			    //storing the pixels as 1d images
			    pixel_RGB *Pixel = (pixel_RGB*)malloc((img_ht)*(img_wd)*sizeof(pixel_RGB));
			    
			    int pix_cnt=0, cnt=0;

			    getline(infile,line); //this stores max value
			    istringstream iss3(line);
			    iss3>>word;

			    max_pixel_val=word;//max pixel value
			    // cout<<max_pixel_val<<endl;
			    unsigned int val;

			    while (getline(infile, line))
			    {
			    	istringstream iss4(line);
			    	for (int i=0; i<=line.length();i++)
			    	{
			    		if(pix_cnt<img_ht*img_wd)
			    		{
			    			val =((int)line[i]);
			    			if(cnt%3==0)
			    			{
			    				Pixel[pix_cnt].r=val;
			    			}
			    			else if(cnt%3==1)
			    			{
			    				Pixel[pix_cnt].g=val;
			    			}
			    			else
			    			{
			    				Pixel[pix_cnt].b=val;
			    				pix_cnt++;
			    			}
			    			cnt++;
			    		}
			    	}
			    }
			    hipEventRecord(stop);
			    hipEventSynchronize(stop);

			    float milliseconds=0;
					hipEventElapsedTime(&milliseconds, start, stop);//get the time in milliseconds


					cout<<"File read in "<<milliseconds<<" ms"<<endl;

			    //COLOR CONVERSION
			    //RGB->XYZ->CIE-L*ab

			    //RGB to XYZ
			    // time_t t9= time(NULL);
					hipEventRecord(start);
					// pixel_XYZ *Pixel_XYZ=RGB_XYZ(Pixel, img_ht, img_wd);
					
			    //XYZ TO CIE-L*ab
					pixel_XYZ* Pixel_LAB=RGB_LAB(Pixel, img_ht, img_wd);
					hipEventRecord(stop);
					hipEventSynchronize(stop);

			    hipEventElapsedTime(&milliseconds, start, stop);//get the time in milliseconds

			    cout<<"Colorspace conversion done in "<<milliseconds<<" ms"<<endl;
			    //IMPLEMENTING SLIC ALGORITHM
			    int N = img_ht*img_wd;  //number of pixels in the images
			    int K = atoi(argv[2]);    //number of superpixels desired

			    int S= floor(sqrt(N/K));//size of each superpixel
			    int m=atoi(argv[3]);    //compactness control constant
			    int k1=ceil(img_ht*1.0/S)*ceil(img_wd*1.0/S);//actual number of superpixels

			    cout<<"Image size: "<<img_wd<<" x "<<img_ht<<endl;
			    cout<<"Using SLIC algorithm to get "<<k1<<" superpixels of approximate size "<<S<<" x "<<S<<", area "<<S*S<<" each, also m/S="<<1.0*m/S<<endl;
			    // cout<<k1<<" "<<S<<" "<<float(img_ht*1.0/S)<<" "<<float(img_wd*1.0/S)<<endl;
			    point* centers_curr=(point*)malloc(k1*sizeof(point));

			    //initialize centers

			    hipEventRecord(start);

			    int center_ctr=0;
			    for(int j=S/2;j<S*ceil(img_ht*1.0/S);j=j+S)
			    {
			    	for(int i=S/2;i<S*ceil(img_wd*1.0/S);i=i+S)
			    	{
			    		int val1=((i>=img_wd)?(img_wd+j-S)/2:i);
			    		int val2=((j>=img_ht)?(img_ht+i-S)/2:j);
			    		centers_curr[center_ctr].x=val1;
			    		centers_curr[center_ctr].y=val2;
			        // cout<<center_ctr<<" "<<centers_curr[center_ctr].x<<" "<<centers_curr[center_ctr].y<<" "<<val1<<" "<<val2<<endl;
			    		center_ctr++;

			    	}
			    }
			    
			    hipEventRecord(stop);
			    hipEventSynchronize(stop);

			    hipEventElapsedTime(&milliseconds, start, stop);//get the time in milliseconds
			    cout<<"centres initialized in "<<milliseconds<<" ms"<<endl;
			    //perturb centers

			    hipEventRecord(start);
			    double* G=(double*)malloc(N*sizeof(double)); 
			    for(int i=0; i<img_wd;i++)
			    {
			    	for(int j=0; j<img_ht;j++)
			    	{
			    		int index=j*img_wd+i;
			    		double L1, L2, L3, L4, a1, a2, a3, a4, b1, b2, b3, b4;
			    		L1=L2=L3=L4=a1=a2=a3=a4=b1=b2=b3=b4=0;

			     		// cout<<i<<" "<<j<<endl;
			     		// pt1 is point(x+1, y),pt 2 is point(x-1,y),pt3 is point(x,y+1), pt4 is point(x,y-1)

			    		if(i+1<img_wd)
			    			L1=Pixel_LAB[j*img_wd+i+1].x, a1=Pixel_LAB[j*img_wd+i+1].y, b1=Pixel_LAB[j*img_wd+i+1].z;
			    		if(i-1>0)
			    			L2=Pixel_LAB[j*img_wd+i-1].x, a2=Pixel_LAB[j*img_wd+i-1].y, b2=Pixel_LAB[j*img_wd+i-1].z;
			    		if(j+1<img_ht)
			    			L3=Pixel_LAB[(j+1)*img_wd+i].x, a3=Pixel_LAB[(j+1)*img_wd+i].y, b3=Pixel_LAB[(j+1)*img_wd+i].z;
			    		if(j-1>0)
			    			L4=Pixel_LAB[(j-1)*img_wd+i].x, a4=Pixel_LAB[(j-1)*img_wd+i].y, b4=Pixel_LAB[(j-1)*img_wd+i].z;

			    		G[index]=pow(L1-L2,2) + pow(a1-a2,2) + pow(b1-b2,2) + pow(L3-L4,2) + pow(a3-a4,2) + pow(b3-b4,2);
			    	}
			    }
			    
			    // cudaDeviceProp prop;

			    for(int i=0; i<k1;i++)  //for every component
			    {
			    	int x1=centers_curr[i].x-1;
			    	int x2=centers_curr[i].x+1;
			    	int y1=centers_curr[i].y-1;
			    	int y2=centers_curr[i].y+1;

			    	int index = min_index(G, N, x1, x2, y1, y2, img_wd);
			    	centers_curr[i].x=(floor)(index%img_wd);
			    	centers_curr[i].y=(floor)(index/img_wd);

			    } 
			    hipEventRecord(stop);
			    hipEventSynchronize(stop);

			    hipEventElapsedTime(&milliseconds, start, stop);//get the time in milliseconds
			    cout<<"centres perturbed in "<<milliseconds<<" ms"<<endl;
			    
			    ///label initialized to all -1
			    int* labels=(int*)malloc(N*sizeof(int));
			    double* d=(double*)malloc(N*sizeof(double));
			    
			    hipEventRecord(start);	    
			    for(int idx=0;idx<N;idx++)
			    {
			    	labels[idx]=-1; 
			    	d[idx]=60000.00;
			    }
			    hipEventRecord(stop);
			    hipEventSynchronize(stop);

			    hipEventElapsedTime(&milliseconds, start, stop);//get the time in milliseconds
			    
			    cout<<"labels and distance measures initialized in "<<milliseconds<<" ms"<<endl;

			    double error=100;

			    point* centers_prev=(point*)(malloc(k1*sizeof(point)));
			    int epoch=0;

			    point* centers_gpu[2];
			    double* d_gpu[2];
			    int* labels_gpu[2];
			    pixel_XYZ* Pixel_LAB_gpu;
			    HANDLE_ERROR(hipMalloc(&centers_gpu[0], k1*sizeof(point)));

			    HANDLE_ERROR(hipMalloc(&labels_gpu[0], N*sizeof(int)));

			    HANDLE_ERROR(hipMalloc(&Pixel_LAB_gpu, N*sizeof(pixel_XYZ)));
			    HANDLE_ERROR(hipMalloc(&d_gpu[0], N*sizeof(double)));


			    HANDLE_ERROR(hipMalloc(&centers_gpu[1], k1*sizeof(point)));

			    HANDLE_ERROR(hipMalloc(&labels_gpu[1], N*sizeof(int)));


			    HANDLE_ERROR(hipMalloc(&d_gpu[1], N*sizeof(double)));

			    hipDeviceProp_t prop;
			    hipGetDeviceProperties(&prop,0);

			    int thread_block1=prop.maxThreadsPerBlock;


			    dim3 DimGrid1(1+(k1/thread_block1),1,1); 
			    dim3 DimBlock1(thread_block1,1,1);
			hipStream_t stream0, stream1;
			hipStreamCreate(&stream0);
			hipStreamCreate(&stream1);


			    while(error>1)
			    {
			    	cout<<"Epoch number "<<epoch<<endl;

			    	for(int i=0; i<k1;i++)
			    	{
						    centers_prev[i].x=centers_curr[i].x; //saving current centres, before any recalculation
						    centers_prev[i].y=centers_curr[i].y;
						}
						
						HANDLE_ERROR(hipMemcpyAsync(labels_gpu[0], labels, N*sizeof(int), hipMemcpyHostToDevice,stream0));
						HANDLE_ERROR(hipMemcpyAsync(centers_gpu[0], centers_curr, k1*sizeof(point), hipMemcpyHostToDevice,stream0));
						HANDLE_ERROR(hipMemcpyAsync(d_gpu[0], d , N*sizeof(double), hipMemcpyHostToDevice,stream0));

						hipEventRecord(start);

						label_assignment<<<DimGrid1,DimBlock1,0,stream0>>>(labels_gpu[0],Pixel_LAB_gpu,centers_gpu[0],S,img_wd, img_ht,m, d_gpu[0], k1);
						hipStreamSynchronize(stream0);
			    			HANDLE_ERROR(hipMemcpyAsync(Pixel_LAB_gpu, Pixel_LAB, N*sizeof(pixel_XYZ), hipMemcpyHostToDevice,stream0));
						HANDLE_ERROR(hipMemcpyAsync(d, d_gpu[0], N*sizeof(double), hipMemcpyDeviceToHost,stream0));
						//copy back centres, labels, d
						    HANDLE_ERROR(hipMemcpyAsync(centers_curr, centers_gpu[0], k1*sizeof(point), hipMemcpyDeviceToHost,stream0));

						    HANDLE_ERROR(hipMemcpyAsync(labels, labels_gpu[0], N*sizeof(int), hipMemcpyDeviceToHost,stream0));
				
						hipEventRecord(stop);
						hipEventSynchronize(stop);
						hipDeviceSynchronize();

						HANDLE_ERROR(hipMemcpyAsync(labels_gpu[1], labels, N*sizeof(int), hipMemcpyHostToDevice,stream1));
						HANDLE_ERROR(hipMemcpyAsync(centers_gpu[1], centers_curr, k1*sizeof(point), hipMemcpyHostToDevice,stream1));
						HANDLE_ERROR(hipMemcpyAsync(d_gpu[1], d , N*sizeof(double), hipMemcpyHostToDevice,stream1));

					hipStream_t stream1;
			    		hipEventElapsedTime(&milliseconds, start, stop);//get the time in milliseconds
			    		hipStreamCreate(&stream1);
					//HANDLE_ERROR(cudaMemcpy(d, d_gpu, N*sizeof(double), cudaMemcpyDeviceToHost));

			    	//cout<<"reached here"<<endl;
			   hipEventRecord(start);
			    	update_centres<<<DimGrid1,DimBlock1,0,stream1>>>(labels_gpu[1], centers_gpu[1], S, img_wd, img_ht, k1);
			    	hipEventRecord(stop);
			    	hipEventSynchronize(stop);
			    	hipStreamSynchronize(stream1);

			    hipEventElapsedTime(&milliseconds, start, stop);//get the time in milliseconds
			    HANDLE_ERROR(hipMemcpyAsync(d, d_gpu[1], N*sizeof(double), hipMemcpyDeviceToHost,stream1));
			//copy back centres, labels, d
			    HANDLE_ERROR(hipMemcpyAsync(centers_curr, centers_gpu[1], k1*sizeof(point), hipMemcpyDeviceToHost,stream1));

			    HANDLE_ERROR(hipMemcpyAsync(labels, labels_gpu[1], N*sizeof(int), hipMemcpyDeviceToHost,stream1));
				
			  // t2=time(NULL);
			    	cout<<"Distances and labels calculated for all points neighboured to centres in "<<milliseconds<<" ms"<<endl;// in "<<double(t2-t1)<<" secs"<<endl;
			    cout<<"cluster centers updated in "<<milliseconds<<" ms"<<endl;

			    //error calculation
			    hipEventRecord(start);
			    error= error_calculation(centers_curr, centers_prev,k1);
			    hipEventRecord(stop);
			    hipEventSynchronize(stop);

			    hipEventElapsedTime(&milliseconds, start, stop);//get the time in milliseconds
			    
			    cout<<"error = "<<error<<" and is calculated in "<<milliseconds<<" ms"<<endl;
			//for(int i=0; i<N;i++)
			    	//cout<<"i="<<i<<" , label="<<labels[i]<<" , distance="<<d[i]<<endl;
			    //HANDLE_ERROR(cudaFree(labels_gpu));
			   // HANDLE_ERROR(cudaFree(Pixel_LAB_gpu));
			    //HANDLE_ERROR(cudaFree(centers_gpu));
			    //HANDLE_ERROR(cudaFree(d_gpu));
			    epoch++;
			}

				HANDLE_ERROR(hipFree(labels_gpu));
			   HANDLE_ERROR(hipFree(Pixel_LAB_gpu));
			    HANDLE_ERROR(hipFree(centers_gpu));
			    HANDLE_ERROR(hipFree(d_gpu));
			
		//enforcing connectivity
			  //for every pixel, find if it is stray, by analysising labels in all 4 directions
			  //if none is same as the pixel, change it to their
			  // for(int x=0; x<img_wd;x++)
			  // {
			  // 	for(int y=0; y<img_ht; y++)
			  // 	{
			  // 		//for the current pixel, get label
			  // 		int L_0=labels[y*img_wd+x];
			  // 		if(L_0!=labels[max(0,y-1)*img_wd+x] && L_0!=labels[min(y,img_ht)*img_wd+x] && L_0!=labels[y*img_wd+min(x+1,img_wd)] && L_0!=labels[y*img_wd+max(0,x-1)])	//comparing with top pixel
					// 	labels[y*img_wd+x]=labels[max(0,y-1)*img_wd+x];	
			  // 	}
			  // }


			pixel_RGB *rgb=(pixel_RGB*)malloc((img_ht)*(img_wd)*sizeof(pixel_RGB));

	///enforce connectivity
	    //for every point, look into its 4 neighbour labels, if all are same and different from pixel's label, change its label
			for(int x=0; x<img_wd; x++)
			{
				for(int y=0; y<img_ht; y++)
				{
					int L_t=labels[max(y-1,0)*img_wd+x];
					int L_b=labels[min(y+1,img_ht)*img_wd+x];
					int L_r=labels[y*img_wd+max(img_wd,x+1)];
					int L_l=labels[y*img_wd+min(0,x-1)];

					if(L_t==L_b && L_b==L_r && L_r==L_l)
					{
						labels[y*img_wd+x]=L_t;
					}
				}
			}
			//randomly shuffle the labels
			random_shuffle(labels,labels+k1);
			float alpha=0;
			hipEventRecord(start);
			for(int i=0;i<img_ht*img_wd;i++)
			{
				int label_val=labels[i];
			      // cout<<label_val<<endl;

				rgb[i].r=alpha*(21*label_val%255)+ (1-alpha)*Pixel[i].r;
				rgb[i].g=alpha*(47*label_val%255) + (1-alpha)*Pixel[i].g;
				rgb[i].b=alpha*(173*label_val%255) + (1-alpha)*Pixel[i].b;
			}
			  //sobel edge detection
			int valX, valY = 0; 
			int GX [3][3];
			int GY [3][3];

			  //Sobel Horizontal Mask     
			GX[0][0] = 1; GX[0][1] = 0; GX[0][2] = -1; 
			GX[1][0] = 2; GX[1][1] = 0; GX[1][2] = -2;  
			GX[2][0] = 1; GX[2][1] = 0; GX[2][2] = -1;

				//Sobel Vertical Mask   
			GY[0][0] =  1; GY[0][1] = 2; GY[0][2] =   1;    
			GY[1][0] =  0; GY[1][1] = 0; GY[1][2] =   0;    
			GY[2][0] = -1; GY[2][1] =-2; GY[2][2] =  -1;

			double val1;
			for(int i=0;i<img_wd;i++)
			{
				for(int j=0;j<img_ht;j++)
				{
					if(i==0||i==img_wd-1||j==0||j==img_ht-1)
					{
						valX=0;
						valY=0;
					}
					else
					{
						valX=0, valY=0;
						for (int x = -1; x <= 1; x++)
						{
							for (int y = -1; y <= 1; y++)
							{
								valX = valX + labels[i+x+(j+y)*img_wd] * GX[1+x][1+y];
								valY = valY + labels[i+x+(j+y)*img_wd]  * GY[1+x][1+y];
							}
						}

					}
					val1=sqrt(valX*valX + valY*valY);
					if(val1>0)
						{	rgb[j*img_wd+i].r=0;
							rgb[j*img_wd+i].g=0;
							rgb[j*img_wd+i].b=0;
						}
					}
				}



			    //labelling the centers
			  // for(int i=0; i<k1;i++)  
			  // {
			  // 	int x_coord=centers_curr[i].x;
			  // 	int y_coord=centers_curr[i].y;
			  //     // cout<<x_coord<<" "<<y_coord<<endl;
			  // 	for (int x=x_coord-5; x<x_coord+5; x++)
			  // 	{
			  // 		for(int y=y_coord-5; y<y_coord+5; y++)
			  // 		{
			  // 			int idx=img_wd*y_coord + x_coord;
			  //         rgb[idx].r= 0;//NULL;//(unsigned char) 0; 
			  //         rgb[idx].g= 0;//(unsigned char) 0; 
			  //         rgb[idx].b= 0;//(unsigned char) 0;
			  //       }
			  //     }
			  //   }
				hipEventRecord(stop);
				hipEventSynchronize(stop);
			    hipEventElapsedTime(&milliseconds, start, stop);//get the time in milliseconds

			    cout<<"Image prepared in "<<milliseconds<<" ms"<<endl;

			    //OUTPUT STORAGE
			    hipEventRecord(start);
			    ofstream ofs;
			    ofs.open("output_gpu.ppm", ofstream::out);
			    ofs<<"P6\n"<<img_wd<<" "<<img_ht<<"\n"<<max_pixel_val<<"\n";

			    for(int j=0; j <img_ht*img_wd;j++)
			    	ofs<<rgb[j].r<<rgb[j].g<<rgb[j].b;

			    ofs.close();
			    hipEventRecord(stop);
			    hipEventSynchronize(stop);
			    hipEventElapsedTime(&milliseconds, start, stop);//get the time in milliseconds

			    cout<<"Image saved in "<<milliseconds<<" ms"<<endl;
			    return 0;
			}
